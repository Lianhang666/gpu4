#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval start, stop;
#define DataType double
#define NUM_STREAMS 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void start_timer() {
    gettimeofday(&start, NULL);
}

void stop_timer(const char* message) {
    gettimeofday(&stop, NULL);
    double elapsedTime = (stop.tv_sec - start.tv_sec) * 1000.0;
    elapsedTime += (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("%s: %.2f ms\n", message, elapsedTime);
}

int main(int argc, char **argv) {
    int inputLength, segmentSize;
    DataType *hostInput1, *hostInput2, *hostOutput, *resultRef;
    DataType *deviceInput1, *deviceInput2, *deviceOutput;

    // Read input length and segment size
    if (argc > 2) {
        inputLength = atoi(argv[1]);
        segmentSize = atoi(argv[2]);
    } else {
        printf("Usage: %s <input_length> <segment_size>\n", argv[0]);
        return 1;
    }
    printf("The input length is %d, segment size is %d\n", inputLength, segmentSize);

    // 分配页锁定内存
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType));
    resultRef = (DataType*)malloc(inputLength * sizeof(DataType));  // 参考结果可以使用普通内存

    // 初始化输入数组
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() % 100;
        hostInput2[i] = rand() % 100;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    // 分配GPU内存
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // 创建CUDA流
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 计算每个流处理的段大小
    int segmentsPerStream = (inputLength + segmentSize - 1) / segmentSize;
    int threadsPerBlock = 256;

    start_timer();

    // Stage 1: 所有Host to Device传输
    for (int i = 0; i < segmentsPerStream; i++) {
        int offset = i * segmentSize;
        int currentSize = min(segmentSize, inputLength - offset);
        int streamIdx = i % NUM_STREAMS;

        hipMemcpyAsync(deviceInput1 + offset, hostInput1 + offset,
                       currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                       streams[streamIdx]);
        hipMemcpyAsync(deviceInput2 + offset, hostInput2 + offset,
                       currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                       streams[streamIdx]);
    }

    // Stage 2: 所有kernel执行
    for (int i = 0; i < segmentsPerStream; i++) {
        int offset = i * segmentSize;
        int currentSize = min(segmentSize, inputLength - offset);
        int streamIdx = i % NUM_STREAMS;
        int blocksPerGrid = (currentSize + threadsPerBlock - 1) / threadsPerBlock;

        vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[streamIdx]>>>
            (deviceInput1 + offset, deviceInput2 + offset, deviceOutput + offset, currentSize);
    }

    // Stage 3: 所有Device to Host传输
    for (int i = 0; i < segmentsPerStream; i++) {
        int offset = i * segmentSize;
        int currentSize = min(segmentSize, inputLength - offset);
        int streamIdx = i % NUM_STREAMS;

        hipMemcpyAsync(hostOutput + offset, deviceOutput + offset,
                       currentSize * sizeof(DataType), hipMemcpyDeviceToHost,
                       streams[streamIdx]);
    }

    // 同步所有流
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    stop_timer("Total execution time with streams");

    // 验证结果
    bool match = true;
    for (int i = 0; i < inputLength; i++) {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-5) {
            printf("Mismatch at index %d: GPU = %f, CPU = %f\n",
                   i, hostOutput[i], resultRef[i]);
            match = false;
            break;
        }
    }
    printf(match ? "Results match.\n" : "Results do not match.\n");

    // 清理资源
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    // 释放GPU内存
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    // 释放CPU内存
    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    free(resultRef);

    return 0;
}
