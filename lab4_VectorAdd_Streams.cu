#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval start, stop;
#define DataType double
#define NUM_STREAMS 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void start_timer() {
    gettimeofday(&start, NULL);
}

void stop_timer(const char* message) {
    gettimeofday(&stop, NULL);
    double elapsedTime = (stop.tv_sec - start.tv_sec) * 1000.0;
    elapsedTime += (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("%s: %.2f ms\n", message, elapsedTime);
}

int main(int argc, char **argv) {
    int inputLength, segmentSize;
    DataType *hostInput1, *hostInput2, *hostOutput, *resultRef;
    DataType *deviceInput1, *deviceInput2, *deviceOutput;
    
    // 读取输入长度和段大小
    if (argc > 2) {
        inputLength = atoi(argv[1]);
        segmentSize = atoi(argv[2]);
    } else {
        printf("Usage: %s <input_length> <segment_size>\n", argv[0]);
        return 1;
    }
    printf("Input length: %d, Segment size: %d\n", inputLength, segmentSize);

    // 分配页锁定内存（为了异步传输）
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType));
    resultRef = (DataType*)malloc(inputLength * sizeof(DataType));

    // 初始化输入数据
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() % 100;
        hostInput2[i] = rand() % 100;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    // 分配GPU内存
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // 创建streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 使用streams处理数据
    start_timer();
    int threadsPerBlock = 256;
    for (int i = 0; i < inputLength; i += segmentSize * NUM_STREAMS) {
        for (int j = 0; j < NUM_STREAMS && (i + j * segmentSize) < inputLength; j++) {
            int offset = i + j * segmentSize;
            int currentSize = min(segmentSize, inputLength - offset);
            int blocksPerGrid = (currentSize + threadsPerBlock - 1) / threadsPerBlock;

            // 异步数据传输和核函数执行
            hipMemcpyAsync(deviceInput1 + offset, hostInput1 + offset,
                           currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                           streams[j]);
            hipMemcpyAsync(deviceInput2 + offset, hostInput2 + offset,
                           currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                           streams[j]);
            
            vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[j]>>>
                (deviceInput1 + offset, deviceInput2 + offset, 
                 deviceOutput + offset, currentSize);

            hipMemcpyAsync(hostOutput + offset, deviceOutput + offset,
                           currentSize * sizeof(DataType), hipMemcpyDeviceToHost,
                           streams[j]);
        }
    }
    
    // 同步所有streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    stop_timer("Total execution time with streams");

    // 验证结果
    bool match = true;
    for (int i = 0; i < inputLength; i++) {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-5) {
            printf("Mismatch at index %d: GPU = %f, CPU = %f\n", 
                   i, hostOutput[i], resultRef[i]);
            match = false;
            break;
        }
    }
    printf(match ? "Results match.\n" : "Results do not match.\n");

    // 清理
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
    
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    free(resultRef);

    return 0;
}
