#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval start, stop;
#define DataType double
#define NUM_STREAMS 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void start_timer() {
    gettimeofday(&start, NULL);
}

void stop_timer(const char* message) {
    gettimeofday(&stop, NULL);
    double elapsedTime = (stop.tv_sec - start.tv_sec) * 1000.0;
    elapsedTime += (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("%s: %.2f ms\n", message, elapsedTime);
}

int main(int argc, char **argv) {
    int inputLength, segmentSize;
    DataType *hostInput1, *hostInput2, *hostOutput;
    DataType *deviceInput1, *deviceInput2, *deviceOutput;
    hipStream_t streams[NUM_STREAMS];
    
    // Get input length and segment size from command line
    if (argc > 2) {
        inputLength = atoi(argv[1]);
        segmentSize = atoi(argv[2]);
    } else {
        printf("Usage: %s <input_length> <segment_size>\n", argv[0]);
        return 1;
    }
    printf("Input length: %d, Segment size: %d\n", inputLength, segmentSize);

    // 分配页锁定内存
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType));

    // Initialize input arrays
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() / (DataType)RAND_MAX;
        hostInput2[i] = rand() / (DataType)RAND_MAX;
    }

    // Allocate device memory
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // Create CUDA streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Start timing
    start_timer();

    // Process data in segments using multiple streams
    int threadsPerBlock = 256;
    for (int offset = 0; offset < inputLength; offset += segmentSize * NUM_STREAMS) {
        for (int i = 0; i < NUM_STREAMS && (offset + i * segmentSize) < inputLength; i++) {
            int currentOffset = offset + i * segmentSize;
            int currentSize = min(segmentSize, inputLength - currentOffset);
            int blocksPerGrid = (currentSize + threadsPerBlock - 1) / threadsPerBlock;

            // Asynchronous memory transfers and kernel execution
            hipMemcpyAsync(deviceInput1 + currentOffset, hostInput1 + currentOffset,
                           currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                           streams[i]);
            hipMemcpyAsync(deviceInput2 + currentOffset, hostInput2 + currentOffset,
                           currentSize * sizeof(DataType), hipMemcpyHostToDevice,
                           streams[i]);

            vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>
                (deviceInput1 + currentOffset, deviceInput2 + currentOffset,
                 deviceOutput + currentOffset, currentSize);

            hipMemcpyAsync(hostOutput + currentOffset, deviceOutput + currentOffset,
                           currentSize * sizeof(DataType), hipMemcpyDeviceToHost,
                           streams[i]);
        }
    }

    // Synchronize all streams before stopping timer
    hipDeviceSynchronize();
    stop_timer("Total execution time with streams");

    // Cleanup
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);

    return 0;
}
