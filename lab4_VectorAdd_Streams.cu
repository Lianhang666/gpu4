
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval start, stop;
#define DataType double
#define NUM_STREAMS 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void start_timer() {
    gettimeofday(&start, NULL);
}

void stop_timer(const char* message) {
    gettimeofday(&stop, NULL);
    double elapsedTime = (stop.tv_sec - start.tv_sec) * 1000.0;
    elapsedTime += (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("%s: %.2f ms\n", message, elapsedTime);
}

int main(int argc, char **argv) {
    int inputLength;
    DataType *hostInput1, *hostInput2, *hostOutput, *resultRef;
    DataType *deviceInput1, *deviceInput2, *deviceOutput;

    if (argc > 1) {
        inputLength = atoi(argv[1]);
    } else {
        printf("Please provide the input length as an argument.\n");
        return 1;
    }
    printf("The input length is %d\n", inputLength);

    // 使用页锁定内存以支持异步传输
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType), hipHostMallocDefault);
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType), hipHostMallocDefault);
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType), hipHostMallocDefault);
    resultRef = (DataType*)malloc(inputLength * sizeof(DataType));

    // 初始化输入数据
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() % 100;
        hostInput2[i] = rand() % 100;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    // 分配GPU内存
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // 创建CUDA流
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 计算每个流处理的数据大小
    int segmentSize = inputLength / NUM_STREAMS;
    int threadsPerBlock = 256;
    int blocksPerSegment = (segmentSize + threadsPerBlock - 1) / threadsPerBlock;

    start_timer();
    
    // 流水线执行
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * segmentSize;
        // 当前流处理的实际大小（处理最后一个段可能不足segmentSize）
        int currentSize = (i == NUM_STREAMS - 1) ? 
            inputLength - offset : segmentSize;

        // 使用当前流异步传输输入数据
        hipMemcpyAsync(deviceInput1 + offset, 
                       hostInput1 + offset,
                       currentSize * sizeof(DataType), 
                       hipMemcpyHostToDevice, 
                       streams[i]);
        hipMemcpyAsync(deviceInput2 + offset, 
                       hostInput2 + offset,
                       currentSize * sizeof(DataType), 
                       hipMemcpyHostToDevice, 
                       streams[i]);

        // 在当前流上启动核函数
        vecAdd<<<blocksPerSegment, threadsPerBlock, 0, streams[i]>>>
            (deviceInput1 + offset, 
             deviceInput2 + offset, 
             deviceOutput + offset, 
             currentSize);

        // 使用当前流异步传输结果
        hipMemcpyAsync(hostOutput + offset, 
                       deviceOutput + offset,
                       currentSize * sizeof(DataType), 
                       hipMemcpyDeviceToHost, 
                       streams[i]);
    }

    // 等待所有流完成
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    stop_timer("Total execution time with streams");

    // 验证结果
    bool match = true;
    for (int i = 0; i < inputLength; i++) {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-5) {
            printf("Mismatch at index %d: GPU = %f, CPU = %f\n",
                   i, hostOutput[i], resultRef[i]);
            match = false;
            break;
        }
    }
    printf(match ? "Results match.\n" : "Results do not match.\n");

    // 清理资源
    hipHostFree(hostInput1);    // 释放页锁定内存
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    free(resultRef);
    
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
