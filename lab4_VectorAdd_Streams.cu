#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval start, stop;
#define DataType double
#define NUM_STREAMS 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void start_timer() {
    gettimeofday(&start, NULL);
}

void stop_timer(const char* message) {
    gettimeofday(&stop, NULL);
    double elapsedTime = (stop.tv_sec - start.tv_sec) * 1000.0;
    elapsedTime += (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("%s: %.2f ms\n", message, elapsedTime);
}

int main(int argc, char **argv) {
    int inputLength, segmentSize;  // S_seg is segmentSize
    DataType *hostInput1, *hostInput2, *hostOutput, *resultRef;
    DataType *deviceInput1, *deviceInput2, *deviceOutput;

    // Read input length and segment size (S_seg)
    if (argc > 2) {
        inputLength = atoi(argv[1]);
        segmentSize = atoi(argv[2]);
    } else {
        printf("Usage: %s <input_length> <segment_size>\n", argv[0]);
        return 1;
    }
    printf("Input length: %d, Segment size (S_seg): %d\n", inputLength, segmentSize);

    // Allocate page-locked host memory for async operations
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType));
    resultRef = (DataType*)malloc(inputLength * sizeof(DataType));

    // Initialize input arrays
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() % 100;
        hostInput2[i] = rand() % 100;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    // Allocate device memory
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Calculate number of segments and thread configuration
    int numSegments = (inputLength + segmentSize - 1) / segmentSize;
    int threadsPerBlock = 256;

    start_timer();

    // Process data in segments using streams
    for (int i = 0; i < numSegments; i++) {
        int offset = i * segmentSize;
        int currentSize = min(segmentSize, inputLength - offset);
        int streamIdx = i % NUM_STREAMS;
        int blocksPerGrid = (currentSize + threadsPerBlock - 1) / threadsPerBlock;

        // Stage 1: Async memory copy to device
        hipMemcpyAsync(deviceInput1 + offset,
                       hostInput1 + offset,
                       currentSize * sizeof(DataType),
                       hipMemcpyHostToDevice,
                       streams[streamIdx]);
        hipMemcpyAsync(deviceInput2 + offset,
                       hostInput2 + offset,
                       currentSize * sizeof(DataType),
                       hipMemcpyHostToDevice,
                       streams[streamIdx]);

        // Stage 2: Launch kernel
        vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[streamIdx]>>>
            (deviceInput1 + offset,
             deviceInput2 + offset,
             deviceOutput + offset,
             currentSize);

        // Stage 3: Async memory copy back to host
        hipMemcpyAsync(hostOutput + offset,
                       deviceOutput + offset,
                       currentSize * sizeof(DataType),
                       hipMemcpyDeviceToHost,
                       streams[streamIdx]);
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    stop_timer("Total execution time with streams");

    // Verify results
    bool match = true;
    for (int i = 0; i < inputLength; i++) {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-5) {
            printf("Mismatch at index %d: GPU = %f, CPU = %f\n",
                   i, hostOutput[i], resultRef[i]);
            match = false;
            break;
        }
    }
    printf(match ? "Results match.\n" : "Results do not match.\n");

    // Cleanup
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    // Free GPU memory
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    // Free CPU memory
    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    free(resultRef);

    return 0;
}
